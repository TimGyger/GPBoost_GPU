
#include <hip/hip_runtime.h>
/*!
* This file is part of GPBoost a C++ library for combining
*	boosting with Gaussian process and mixed effects models
*
* Copyright (c) 2020 Fabio Sigrist. All rights reserved.
*
* Licensed under the Apache License Version 2.0. See LICENSE file in the project root for license information.
*/
#ifdef USE_CUDA_GP
#include <GPBoost/GP_utils.h>
#include <cuda_runtime.h>
#include <cublas_v2.h>
#include <cusparse.h>
#include <device_launch_parameters.h>
#include <cusolverDn.h>
#include <LightGBM/utils/log.h>
using LightGBM::Log;

namespace GPBoost {

    bool try_matmul_gpu(const den_mat_t& A, const den_mat_t& B, den_mat_t& C) {
        int M = A.rows(), K = A.cols(), N = B.cols();
        if (K != B.rows()) {
            Log::REInfo("[GPU] Dimension mismatch.");
            return false;
        }

        C.resize(M, N);

        const double* h_A = A.data();
        const double* h_B = B.data();
        double* h_C = C.data();

        double* d_A = nullptr, * d_B = nullptr, * d_C = nullptr;
        cudaError_t cuda_stat;
        cublasStatus_t stat;
        cublasHandle_t handle;

        size_t size_A = M * K * sizeof(double);
        size_t size_B = K * N * sizeof(double);
        size_t size_C = M * N * sizeof(double);

        cuda_stat = cudaMalloc((void**)&d_A, size_A);
        if (cuda_stat != cudaSuccess) return false;
        cuda_stat = cudaMalloc((void**)&d_B, size_B);
        if (cuda_stat != cudaSuccess) {
            cudaFree(d_A);
            return false;
        }

        cuda_stat = cudaMalloc((void**)&d_C, size_C);
        if (cuda_stat != cudaSuccess) {
            cudaFree(d_A); cudaFree(d_B);
            return false;
        }

        cudaMemcpy(d_A, h_A, size_A, cudaMemcpyHostToDevice);
        cudaMemcpy(d_B, h_B, size_B, cudaMemcpyHostToDevice);

        stat = cublasCreate(&handle);
        if (stat != CUBLAS_STATUS_SUCCESS) {
            cudaFree(d_A); cudaFree(d_B); cudaFree(d_C);
            return false;
        }

        const double alpha = 1.0;
        const double beta = 0.0;

        // cuBLAS performs: C = alpha * op(A) * op(B) + beta * C
        // We want: C = A * B
        // A: MxK, B: KxN, C: MxN
        // So op(A) = A, op(B) = B
        stat = cublasDgemm(handle,
            CUBLAS_OP_N, CUBLAS_OP_N,  // No transpose
            M, N, K,                   // C is MxN, A is MxK, B is KxN
            &alpha,
            d_A, M,  // lda = leading dim of A = M (since column-major)
            d_B, K,  // ldb = leading dim of B = K
            &beta,
            d_C, M); // ldc = leading dim of C = M

        if (stat != CUBLAS_STATUS_SUCCESS) {
            cublasDestroy(handle);
            cudaFree(d_A); cudaFree(d_B); cudaFree(d_C);
            return false;
        }

        cudaMemcpy(h_C, d_C, size_C, cudaMemcpyDeviceToHost);

        cublasDestroy(handle);
        cudaFree(d_A); cudaFree(d_B); cudaFree(d_C);

        Log::REInfo("[GPU] Matrix multiplication completed with cuBLAS.");
        return true;
    }

    bool try_diag_times_dense_gpu(const vec_t& D, const den_mat_t& B, den_mat_t& C) {
        int M = B.rows();
        int N = B.cols();

        if (D.size() != M) {
            Log::REInfo("[GPU] Dimension mismatch between diagonal and matrix.");
            return false;
        }

        C.resize(M, N);

        // Host pointers
        const double* h_D = D.data();
        const double* h_B = B.data();
        double* h_C = C.data();

        // Device pointers
        double* d_D = nullptr;
        double* d_B = nullptr;
        double* d_C = nullptr;

        cudaMalloc((void**)&d_D, M * sizeof(double));
        cudaMalloc((void**)&d_B, M * N * sizeof(double));
        cudaMalloc((void**)&d_C, M * N * sizeof(double));

        cudaMemcpy(d_D, h_D, M * sizeof(double), cudaMemcpyHostToDevice);
        cudaMemcpy(d_B, h_B, M * N * sizeof(double), cudaMemcpyHostToDevice);
        // Create cuBLAS handle
        cublasHandle_t handle;
        cublasCreate(&handle);
        // Multiply: C = diag(D) * B (i.e., scale each row of B by D[i])
        // Use cuBLAS: d_C = diag(d_D) * d_B
        cublasStatus_t stat = cublasDdgmm(handle,
            CUBLAS_SIDE_LEFT, // Left = scale rows (use RIGHT to scale columns)
            M, N,
            d_B, M,
            d_D, 1, // stride = 1
            d_C, M);
        if (stat != CUBLAS_STATUS_SUCCESS) {
            Log::REInfo("[GPU] cuBLAS Ddgmm failed.");
            cudaFree(d_D); cudaFree(d_B); cudaFree(d_C);
            cublasDestroy(handle);
            return false;
        }

        cudaMemcpy(h_C, d_C, M * N * sizeof(double), cudaMemcpyDeviceToHost);

        // Clean up
        cudaFree(d_D);
        cudaFree(d_B);
        cudaFree(d_C);
        cublasDestroy(handle);

        Log::REInfo("[GPU] Diagonal x Dense matrix multiplication completed with cuBLAS.");
        return true;
    }

    bool try_spmatmul_gpu(const sp_mat_rm_t& A, const sp_mat_rm_t& B, sp_mat_rm_t& C) {
        if (A.cols() != B.rows()) return false;

        cudaError_t cuda_stat;
        cusparseStatus_t cusparse_stat;
        cusparseHandle_t handle = nullptr;
        cusparseSpMatDescr_t matA = nullptr, matB = nullptr, matC = nullptr;
        cusparseSpGEMMDescr_t spgemmDescr = nullptr;

        int m = A.rows(), k = A.cols(), n = B.cols();
        int A_nnz = A.nonZeros(), B_nnz = B.nonZeros();

        int* d_A_rowPtr = nullptr, * d_A_colInd = nullptr;
        double* d_A_values = nullptr;
        int* d_B_rowPtr = nullptr, * d_B_colInd = nullptr;
        double* d_B_values = nullptr;
        int* d_C_rowPtr = nullptr, * d_C_colInd = nullptr;
        double* d_C_values = nullptr;
        void* dBuffer1 = nullptr, * dBuffer2 = nullptr;

        // Allocate device memory for A
        cudaMalloc(&d_A_rowPtr, (m + 1) * sizeof(int));
        cudaMalloc(&d_A_colInd, A_nnz * sizeof(int));
        cudaMalloc(&d_A_values, A_nnz * sizeof(double));

        // Allocate device memory for B
        cudaMalloc(&d_B_rowPtr, (k + 1) * sizeof(int));
        cudaMalloc(&d_B_colInd, B_nnz * sizeof(int));
        cudaMalloc(&d_B_values, B_nnz * sizeof(double));

        // Copy A and B to device
        cudaMemcpy(d_A_rowPtr, A.outerIndexPtr(), (m + 1) * sizeof(int), cudaMemcpyHostToDevice);
        cudaMemcpy(d_A_colInd, A.innerIndexPtr(), A_nnz * sizeof(int), cudaMemcpyHostToDevice);
        cudaMemcpy(d_A_values, A.valuePtr(), A_nnz * sizeof(double), cudaMemcpyHostToDevice);

        cudaMemcpy(d_B_rowPtr, B.outerIndexPtr(), (k + 1) * sizeof(int), cudaMemcpyHostToDevice);
        cudaMemcpy(d_B_colInd, B.innerIndexPtr(), B_nnz * sizeof(int), cudaMemcpyHostToDevice);
        cudaMemcpy(d_B_values, B.valuePtr(), B_nnz * sizeof(double), cudaMemcpyHostToDevice);

        // cuSPARSE setup
        cusparseCreate(&handle);
        //cusparseCreateSpGEMMDescr(&spgemmDesc);
        cusparseSpGEMM_createDescr(&spgemmDescr);
        cusparseCreateCsr(&matA, m, k, A_nnz, d_A_rowPtr, d_A_colInd, d_A_values,
            CUSPARSE_INDEX_32I, CUSPARSE_INDEX_32I, CUSPARSE_INDEX_BASE_ZERO, CUDA_R_64F);

        cusparseCreateCsr(&matB, k, n, B_nnz, d_B_rowPtr, d_B_colInd, d_B_values,
            CUSPARSE_INDEX_32I, CUSPARSE_INDEX_32I, CUSPARSE_INDEX_BASE_ZERO, CUDA_R_64F);

        cusparseCreateCsr(&matC, m, n, 0, nullptr, nullptr, nullptr,
            CUSPARSE_INDEX_32I, CUSPARSE_INDEX_32I, CUSPARSE_INDEX_BASE_ZERO, CUDA_R_64F);

        double alpha = 1.0, beta = 0.0;
        size_t bufferSize1 = 0, bufferSize2 = 0;

        // Phase 1: Work estimation
        cusparseSpGEMM_workEstimation(handle, CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, CUDA_R_64F,
            CUSPARSE_SPGEMM_DEFAULT, spgemmDescr, &bufferSize1, nullptr);
        cudaMalloc(&dBuffer1, bufferSize1);
        cusparseSpGEMM_workEstimation(handle, CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, CUDA_R_64F,
            CUSPARSE_SPGEMM_DEFAULT, spgemmDescr, &bufferSize1, dBuffer1);

        // Phase 2: Compute
        cusparseSpGEMM_compute(handle, CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, CUDA_R_64F,
            CUSPARSE_SPGEMM_DEFAULT, spgemmDescr, &bufferSize2, nullptr);
        cudaMalloc(&dBuffer2, bufferSize2);
        cusparseSpGEMM_compute(handle, CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, CUDA_R_64F,
            CUSPARSE_SPGEMM_DEFAULT, spgemmDescr, &bufferSize2, dBuffer2);

        // Phase 3: Copy to finalize matC
        int64_t C_num_rows, C_num_cols, C_nnz;
        cusparseSpMatGetSize(matC, &C_num_rows, &C_num_cols, &C_nnz);
        cudaMalloc(&d_C_rowPtr, (m + 1) * sizeof(int));
        cudaMalloc(&d_C_colInd, C_nnz * sizeof(int));
        cudaMalloc(&d_C_values, C_nnz * sizeof(double));

        cusparseCsrSetPointers(matC, d_C_rowPtr, d_C_colInd, d_C_values);
        cusparseSpGEMM_copy(handle, CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, CUDA_R_64F,
            CUSPARSE_SPGEMM_DEFAULT, spgemmDescr);

        // Copy result to host
        std::vector<int> h_C_rowPtr(m + 1);
        std::vector<int> h_C_colInd(C_nnz);
        std::vector<double> h_C_values(C_nnz);

        cudaMemcpy(h_C_rowPtr.data(), d_C_rowPtr, (m + 1) * sizeof(int), cudaMemcpyDeviceToHost);
        cudaMemcpy(h_C_colInd.data(), d_C_colInd, C_nnz * sizeof(int), cudaMemcpyDeviceToHost);
        cudaMemcpy(h_C_values.data(), d_C_values, C_nnz * sizeof(double), cudaMemcpyDeviceToHost);

        // Build result Eigen matrix
        C.resize(m, n);
        C.makeCompressed();
        C.reserve(C_nnz);
        std::copy(h_C_rowPtr.begin(), h_C_rowPtr.end(), C.outerIndexPtr());
        std::copy(h_C_colInd.begin(), h_C_colInd.end(), C.innerIndexPtr());
        std::copy(h_C_values.begin(), h_C_values.end(), C.valuePtr());

        // Cleanup
        cudaFree(d_A_rowPtr); cudaFree(d_A_colInd); cudaFree(d_A_values);
        cudaFree(d_B_rowPtr); cudaFree(d_B_colInd); cudaFree(d_B_values);
        cudaFree(d_C_rowPtr); cudaFree(d_C_colInd); cudaFree(d_C_values);
        cudaFree(dBuffer1); cudaFree(dBuffer2);
        cusparseDestroySpMat(matA); cusparseDestroySpMat(matB); cusparseDestroySpMat(matC);
        //cusparseDestroySpGEMMDescr(spgemmDesc);
        cusparseSpGEMM_destroyDescr(spgemmDescr);
        cusparseDestroy(handle);

        return true;
    }

    bool try_sparse_dense_matmul_gpu(const sp_mat_rm_t& A, const den_mat_t& B, den_mat_t& C) {
        int M = A.rows(), K = A.cols(), N = B.cols();
        if (K != B.rows()) {
            Log::REInfo("[GPU] Dimension mismatch.");
            return false;
        }

        const int nnz = A.nonZeros();
        const int* h_csrOffsets = A.outerIndexPtr();  // Row pointers
        const int* h_columns = A.innerIndexPtr();     // Column indices
        const double* h_values = A.valuePtr();        // Non-zero values

        int* d_csrOffsets = nullptr;
        int* d_columns = nullptr;
        double* d_values = nullptr;
        double* d_B = nullptr;
        double* d_C = nullptr;

        cudaMalloc((void**)&d_csrOffsets, (M + 1) * sizeof(int));
        cudaMalloc((void**)&d_columns, nnz * sizeof(int));
        cudaMalloc((void**)&d_values, nnz * sizeof(double));
        cudaMalloc((void**)&d_B, K * N * sizeof(double));
        cudaMalloc((void**)&d_C, M * N * sizeof(double));

        cudaMemcpy(d_csrOffsets, h_csrOffsets, (M + 1) * sizeof(int), cudaMemcpyHostToDevice);
        cudaMemcpy(d_columns, h_columns, nnz * sizeof(int), cudaMemcpyHostToDevice);
        cudaMemcpy(d_values, h_values, nnz * sizeof(double), cudaMemcpyHostToDevice);
        cudaMemcpy(d_B, B.data(), K * N * sizeof(double), cudaMemcpyHostToDevice);

        cusparseHandle_t handle;
        cusparseCreate(&handle);

        cusparseSpMatDescr_t matA;
        cusparseDnMatDescr_t matB, matC;

        cusparseCreateCsr(&matA, M, K, nnz,
            d_csrOffsets, d_columns, d_values,
            CUSPARSE_INDEX_32I, CUSPARSE_INDEX_32I,
            CUSPARSE_INDEX_BASE_ZERO, CUDA_R_64F);

        cusparseCreateDnMat(&matB, K, N, K, d_B, CUDA_R_64F, CUSPARSE_ORDER_COL);
        cusparseCreateDnMat(&matC, M, N, M, d_C, CUDA_R_64F, CUSPARSE_ORDER_COL);

        const double alpha = 1.0;
        const double beta = 0.0;

        size_t bufferSize = 0;
        void* dBuffer = nullptr;
        cusparseSpMM_bufferSize(handle,
            CUSPARSE_OPERATION_NON_TRANSPOSE,
            CUSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC,
            CUDA_R_64F, CUSPARSE_SPMM_CSR_ALG2,
            &bufferSize);

        cudaMalloc(&dBuffer, bufferSize);

        cusparseStatus_t stat = cusparseSpMM(handle,
            CUSPARSE_OPERATION_NON_TRANSPOSE,
            CUSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC,
            CUDA_R_64F, CUSPARSE_SPMM_CSR_ALG2,
            dBuffer);

        if (stat != CUSPARSE_STATUS_SUCCESS) {
            Log::REInfo("[GPU] cuSPARSE SpMM failed.");
            cudaFree(dBuffer); cudaFree(d_csrOffsets); cudaFree(d_columns);
            cudaFree(d_values); cudaFree(d_B); cudaFree(d_C);
            cusparseDestroySpMat(matA); cusparseDestroyDnMat(matB);
            cusparseDestroyDnMat(matC); cusparseDestroy(handle);
            return false;
        }

        C.resize(M, N);  // Resize Eigen matrix before copying
        cudaMemcpy(C.data(), d_C, M * N * sizeof(double), cudaMemcpyDeviceToHost);

        // Clean up
        cudaFree(dBuffer); cudaFree(d_csrOffsets); cudaFree(d_columns);
        cudaFree(d_values); cudaFree(d_B); cudaFree(d_C);
        cusparseDestroySpMat(matA); cusparseDestroyDnMat(matB);
        cusparseDestroyDnMat(matC); cusparseDestroy(handle);

        return true;
    }

    bool try_solve_lower_triangular_gpu(const chol_den_mat_t& chol, const den_mat_t& R_host, den_mat_t& X_host) {
        den_mat_t L_host = chol.matrixL();
        int n = L_host.rows();
        int m = R_host.cols();
        if (L_host.cols() != n || R_host.rows() != n) {
            return false;
        }
        X_host.resize(n, m);
        // Allocate device memory
        double* d_L = nullptr;
        double* d_X = nullptr;

        cudaMalloc(&d_L, n * n * sizeof(double));
        cudaMalloc(&d_X, n * m * sizeof(double));

        cudaMemcpy(d_L, L_host.data(), n * n * sizeof(double), cudaMemcpyHostToDevice);
        cudaMemcpy(d_X, R_host.data(), n * m * sizeof(double), cudaMemcpyHostToDevice);

        // Create cuBLAS handle
        cublasHandle_t handle;
        cublasStatus_t stat = cublasCreate(&handle);
        if (stat != CUBLAS_STATUS_SUCCESS) {
            cudaFree(d_L); cudaFree(d_X);
            return false;
        }
        const double alpha = 1.0;

        // Solve: L * X = R -> X = L^{-1} * R
        // L is lower-triangular, column-major
        // Left-side, lower-triangular, no transpose, non-unit diagonal
        stat = cublasDtrsm(
            handle,
            CUBLAS_SIDE_LEFT,      // Solve L * X = R
            CUBLAS_FILL_MODE_LOWER,
            CUBLAS_OP_N,           // No transpose
            CUBLAS_DIAG_NON_UNIT,  // Assume general diagonal
            n,                     // number of rows of L and X
            m,                     // number of columns of X
            &alpha,                // Scalar alpha
            d_L, n,                // L, leading dimension n
            d_X, n                 // R becomes X, leading dimension n
        );

        if (stat != CUBLAS_STATUS_SUCCESS) {
            cudaFree(d_L); cudaFree(d_X);
            cublasDestroy(handle);
            return false;
        }

        // Copy result back
        cudaMemcpy(X_host.data(), d_X, n * m * sizeof(double), cudaMemcpyDeviceToHost);

        // Cleanup
        cudaFree(d_L);
        cudaFree(d_X);
        cublasDestroy(handle);

        Log::REInfo("[GPU] Triangular solve with CUBLAS.");
        return true;
    }

    

    // CUDA kernel: Sigma(i,j) -= dot(M1.col(i), M2.col(j))
    __global__ void subtract_prod_from_mat_kernel(
        const double* __restrict__ M1,
        const double* __restrict__ M2,
        double* Sigma,
        int M1_rows, int M1_cols,
        int M2_rows, int M2_cols,
        bool only_triangular)
    {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;

        if (i >= M1_cols || j >= M2_cols) return;
        if (only_triangular && j < i) return;

        double dot = 0.0;
        for (int k = 0; k < M1_rows; ++k) {
            dot += M1[i * M1_rows + k] * M2[j * M2_rows + k];
        }

        // column-major access: Sigma(i, j) => j * rows + i
        atomicAdd(&Sigma[j * M1_cols + i], -dot);

        if (!only_triangular && j > i) {
            atomicAdd(&Sigma[i * M1_cols + j], -dot);  // symmetric fill
        }
    }
    __global__ void subtract_prod_from_sparse_mat_kernel(
    const int* __restrict__ row_ptr,
    const int* __restrict__ col_idx,
    double* __restrict__ values,
    const double* __restrict__ M1,  // Shape: (n_rows, K)
    const double* __restrict__ M2,  // Shape: (n_cols, K)
    int n_rows, int n_cols, int K)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n_rows) return;

    int row_start = row_ptr[row];
    int row_end = row_ptr[row + 1];

    for (int idx = row_start; idx < row_end; ++idx) {
        int col = col_idx[idx];

        // Only compute upper triangle or diagonal
        if (row <= col) {
            double dot = 0.0;
            for (int k = 0; k < K; ++k) {
                dot += M1[row * K + k] * M2[col * K + k];
            }
            atomicAdd(&values[idx], -dot);
        }
            // Note: for full symmetry, the host must mirror Sigma(j,i) = Sigma(i,j) afterwards
    }
}

    void launch_subtract_sparse_kernel(
        const int* row_ptr, const int* col_idx, double* values,
        const double* M1, const double* M2,
        int n, int m, int K, bool only_triangular)
    {
        int blockSize = 256;
        int numBlocks = (n + blockSize - 1) / blockSize;
        subtract_prod_from_sparse_mat_kernel << <numBlocks, blockSize >> > (
            row_ptr, col_idx, values, M1, M2, n, m, K);
    }

    void launch_subtract_prod_from_mat_kernel(
        const double* M1, const double* M2, double* Sigma,
        int M1_rows, int M1_cols,
        int M2_rows, int M2_cols,
        bool only_triangular)
    {
        dim3 blockDim(16, 16);
        dim3 gridDim((M2_cols + blockDim.x - 1) / blockDim.x,
            (M1_cols + blockDim.y - 1) / blockDim.y);

        subtract_prod_from_mat_kernel << <gridDim, blockDim >> > (
            M1, M2, Sigma,
            M1_rows, M1_cols,
            M2_rows, M2_cols,
            only_triangular
            );
        cudaDeviceSynchronize();
    }

    
    bool cholesky_cusolver_to_eigen(chol_den_mat_t& llt, const den_mat_t& A_input) {
        int N = A_input.rows();
        if (A_input.cols() != N) {
            Log::REInfo("Input matrix is not square.");
            return false;
        }

        // Step 1: Create cuSolver handle
        cusolverDnHandle_t handle;
        cusolverStatus_t status = cusolverDnCreate(&handle);
        if (status != CUSOLVER_STATUS_SUCCESS) {
            Log::REInfo("cuSOLVER initialization failed.");
            return false;
        }

        // Step 2: Allocate GPU memory for matrix
        double* d_A = nullptr;
        cudaError_t cudaStat = cudaMalloc(&d_A, sizeof(double) * N * N);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMalloc failed for d_A");
            cusolverDnDestroy(handle);
            return false;
        }

        cudaStat = cudaMemcpy(d_A, A_input.data(), sizeof(double) * N * N, cudaMemcpyHostToDevice);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMemcpy failed");
            cudaFree(d_A);
            cusolverDnDestroy(handle);
            return false;
        }

        // Step 3: Query buffer size
        int work_size = 0;
        status = cusolverDnDpotrf_bufferSize(handle, CUBLAS_FILL_MODE_LOWER, N, d_A, N, &work_size);
        if (status != CUSOLVER_STATUS_SUCCESS) {
            Log::REInfo("cusolverDnDpotrf_bufferSize failed.");
            cudaFree(d_A);
            cusolverDnDestroy(handle);
            return false;
        }

        double* work = nullptr;
        cudaStat = cudaMalloc(&work, sizeof(double) * work_size);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMalloc failed for workspace");
            cudaFree(d_A);
            cusolverDnDestroy(handle);
            return false;
        }

        int* dev_info = nullptr;
        cudaStat = cudaMalloc(&dev_info, sizeof(int));
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMalloc failed ");
            cudaFree(d_A);
            cudaFree(work);
            cusolverDnDestroy(handle);
            return false;
        }

        // Step 4: Compute Cholesky factorization
        status = cusolverDnDpotrf(handle, CUBLAS_FILL_MODE_LOWER, N, d_A, N, work, work_size, dev_info);
        if (status != CUSOLVER_STATUS_SUCCESS) {
            Log::REInfo("cusolverDnDpotrf failed.");
            cudaFree(d_A); cudaFree(work); cudaFree(dev_info);
            cusolverDnDestroy(handle);
            return false;
        }

        int dev_info_h = 0;
        cudaStat = cudaMemcpy(&dev_info_h, dev_info, sizeof(int), cudaMemcpyDeviceToHost);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMemcpy failed");
            cudaFree(d_A); cudaFree(work); cudaFree(dev_info);
            cusolverDnDestroy(handle);
            return false;
        }

        if (dev_info_h != 0) {
            Log::REInfo("Cholesky factorization failed on GPU");
            cudaFree(d_A); cudaFree(work); cudaFree(dev_info);
            cusolverDnDestroy(handle);
            return false;
        }

        // Step 5: Copy result back (only lower triangle)
        den_mat_t L(N, N);
        cudaStat = cudaMemcpy(L.data(), d_A, sizeof(double) * N * N, cudaMemcpyDeviceToHost);
        if (cudaStat != cudaSuccess) {
            Log::REInfo("cudaMemcpy failed");
            cudaFree(d_A); cudaFree(work); cudaFree(dev_info);
            cusolverDnDestroy(handle);
            return false;
        }

        // Step 6: Feed to Eigen's LLT (only lower triangle will be used)
        llt.compute(L.selfadjointView<Eigen::Lower>());

        // Step 7: Cleanup
        cudaFree(d_A);
        cudaFree(work);
        cudaFree(dev_info);
        cusolverDnDestroy(handle);

        Log::REInfo("[GPU] Cholesky factorization with cuSOLVER completed successfully.");
        return true;
    }

}  // namespace GPBoost

#endif  // USE_CUDA_GP
